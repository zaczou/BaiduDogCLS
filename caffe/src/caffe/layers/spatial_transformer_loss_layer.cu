#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/spatial_transformer_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void STLossForwardGPU(const int nthreads, int num_, 
		int output_h_, int output_w_, const Dtype* theta, Dtype* loss_array) {
	
	CUDA_KERNEL_LOOP(index, nthreads) {

		const int t = index % output_w_;
		const int s = (index / output_w_) % output_h_;
		const int i = index / (output_w_ * output_h_);

		Dtype input_x = s * 2.0 / output_h_ - 1;
		Dtype input_y = t * 2.0 / output_w_ - 1;
		
		Dtype output_x = theta[6*i] * input_x + theta[6*i+1] * input_y + theta[6*i+2];
		Dtype output_y = theta[6*i+3] * input_x + theta[6*i+4] * input_y + theta[6*i+5];
		
		Dtype loss = (Dtype)0;
		
		if(output_x < -1) {
			loss += (output_x + 1) * (output_x + 1) / 2;
		} else if(output_x > 1) {
			loss += (output_x - 1) * (output_x - 1) / 2;
		}
		
		if(output_y < -1) {
			loss += (output_y + 1) * (output_y + 1) / 2;
		} else if(output_y > 1) {
			loss += (output_y - 1) * (output_y - 1) / 2;
		}
		
		loss_array[index] = loss;
  }
}

template <typename Dtype>
void STLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
	
	string prefix = "STLossLayer::Forward_gpu::\t";

	const Dtype* theta = bottom[0]->gpu_data();
	Dtype* loss_array = loss_.mutable_gpu_data();
	
	caffe_gpu_set(loss_.count(), (Dtype)0, loss_array);
	
	const int nthreads = num_ * output_h_ * output_w_;
	STLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
	     CAFFE_CUDA_NUM_THREADS>>>(nthreads, num_, output_h_, output_w_, theta, loss_array);
	
	Dtype loss;
	caffe_gpu_asum(nthreads, loss_array, &loss);
	loss /= nthreads;
	
	top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void STLossBackwardGPU(const int nthreads, int num_, 
		int output_h_, int output_w_, const Dtype* theta, Dtype* dtheta_tmp) {
	
	CUDA_KERNEL_LOOP(index, nthreads) {

		const int t = index % output_w_;
		const int s = (index / output_w_) % output_h_;
		const int i = index / (output_w_ * output_h_);

		Dtype input_x = s * 2.0 / output_h_ - 1;
		Dtype input_y = t * 2.0 / output_w_ - 1;
		
		Dtype output_x = theta[6*i] * input_x + theta[6*i+1] * input_y + theta[6*i+2];
		Dtype output_y = theta[6*i+3] * input_x + theta[6*i+4] * input_y + theta[6*i+5];
		
		Dtype d1 = (Dtype)0, d2 = (Dtype)0;
		
		if(output_x < -1) {
			d1 = output_x + 1;
		} else if(output_x > 1) {
			d1 = output_x - 1;
		}
		
		if(output_y < -1) {
			d2 = output_y + 1;
		} else if(output_y > 1) {
			d2 = output_y - 1;
		}
		
		dtheta_tmp[(6*i) * (output_h_ * output_w_) + s * output_w_ + t] = d1 * input_x;
		dtheta_tmp[(6*i+1) * (output_h_ * output_w_) + s * output_w_ + t] = d1 * input_y;
		dtheta_tmp[(6*i+2) * (output_h_ * output_w_) + s * output_w_ + t] = d1;
		dtheta_tmp[(6*i+3) * (output_h_ * output_w_) + s * output_w_ + t] = d2 * input_x;
		dtheta_tmp[(6*i+4) * (output_h_ * output_w_) + s * output_w_ + t] = d2 * input_y;
		dtheta_tmp[(6*i+5) * (output_h_ * output_w_) + s * output_w_ + t] = d2;
  }
}

template <typename Dtype>
void STLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	
	const Dtype* theta = bottom[0]->gpu_data();
	Dtype* dtheta_tmp = dtheta_tmp_.mutable_gpu_data();
	Dtype* all_ones_vec = all_ones_vec_.mutable_gpu_data();
	Dtype* dtheta = bottom[0]->mutable_gpu_diff();
	
	caffe_gpu_set(all_ones_vec_.count(), (Dtype)1, all_ones_vec);
	
	const int nthreads = num_ * output_h_ * output_w_;
	STLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
	     CAFFE_CUDA_NUM_THREADS>>>(nthreads, num_, output_h_, output_w_, theta, dtheta_tmp);
	     
	caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * 6, 1, output_h_ * output_w_, 
			(Dtype)1., dtheta_tmp, all_ones_vec, (Dtype)0., dtheta);
			
	caffe_gpu_scal(bottom[0]->count(), top[0]->cpu_diff()[0] / nthreads, dtheta);
}

INSTANTIATE_LAYER_GPU_FUNCS(STLossLayer);

}  // namespace caffe